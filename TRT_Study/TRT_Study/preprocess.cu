#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdint.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

using namespace std;

// ��ó�� �Լ� 0 (NHWC->NCHW, BGR->RGB, [0, 255]->[0.0, 1.0](Normalize))
__global__ void kernel_preprocess_0(
    float* output,				// [N,RGB,H,W]
    const unsigned char* input, // [N,H,W,BGR]
    const int batchSize, const int height, const int width, const int channel,
    const int tcount)
{
    int pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (pos >= tcount) return;

    const int w_idx = pos % width;
    int idx = pos / width;
    const int h_idx = idx % height;
    idx /= height;
    const int c_idx = idx % channel;
    const int b_idx = idx / channel;

    int g_idx = b_idx * height * width * channel + h_idx * width * channel + w_idx * channel + 2 - c_idx;

    output[pos] = input[g_idx] / 255.f;
}

void preprocess_cu_0(float* output, unsigned char* input, int batchSize, int height, int width, int channel, hipStream_t stream)
{
    int tcount = batchSize * height * width * channel;
    int block = 512;
    int grid = (tcount - 1) / block + 1;

    kernel_preprocess_0 << <grid, block, 0, stream >> > (output, input, batchSize, height, width, channel, tcount);
}

// ��ó�� �Լ� 1 (NHWC->NCHW, BGR->RGB, [0, 255]->[0.0, 1.0], 
// Normalize(mean = [0.485, 0.456, 0.406], std = [0.229, 0.224, 0.225]))
__constant__ float constMem_mean_std[6];
__global__ void kernel_preprocess_1(
    float* output,				// [N,RGB,H,W]
    const unsigned char* input, // [N,H,W,BGR]
    const int batchSize, const int height, const int width, const int channel,
    const int tcount)
{
    int pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (pos >= tcount) return;

    const int w_idx = pos % width;
    int idx = pos / width;
    const int h_idx = idx % height;
    idx /= height;
    const int c_idx = idx % channel;
    const int b_idx = idx / channel;

    int g_idx = b_idx * height * width * channel + h_idx * width * channel + w_idx * channel + 2 - c_idx;

    output[pos] = (input[g_idx] / 255.f - constMem_mean_std[c_idx]) / constMem_mean_std[c_idx + 3];
}

void preprocess_cu_1(float* output, unsigned char* input, int batchSize, int height, int width, int channel, std::vector<float>& mean_std, hipStream_t stream)
{
    int tcount = batchSize * height * width * channel;
    int block = 512;
    int grid = (tcount - 1) / block + 1;

    //hipEvent_t start, stop;
    //hipEventCreate(&start);
    //hipEventCreate(&stop);
    //hipEventRecord(start);
    hipMemcpyToSymbol(HIP_SYMBOL(constMem_mean_std), mean_std.data(), sizeof(float) * 6);
    kernel_preprocess_1 << <grid, block, 0, stream >> > (output, input, batchSize, height, width, channel, tcount);
    //hipEventRecord(stop);
    //hipEventSynchronize(stop);
    //float time;
    //hipEventElapsedTime(&time, start, stop);
    //std::cout << "elapsed time :: " << time << std::endl;
    //hipEventDestroy(start);
    //hipEventDestroy(stop);
    //elapsed time :: 0.635904 
    //elapsed time :: 0.599040 (cuda constant mem w data transfer)
    //elapsed time :: 0.492544 (cuda constant mem wo data transfer)

}

__device__ __forceinline__ float area_pixel_compute_source_index(float scale, int dst_index)
{
    float src_idx = scale * (dst_index + 0.5f) - 0.5f;
    return src_idx < 0 ? 0 : src_idx;
}
// ��ó�� �Լ� 3 (NHWC->NCHW, BGR->RGB, [0, 255]->[0.0, 1.0](Normalize), letterbox padding)
__global__ void kernel_preprocess_3(
    float* output,              // [N,RGB,P,Q]
    const unsigned char* input, // [N,H,W,BGR]
    float rheight, float rwidth,
    int P, int Q,   //OUTPUT(padded) for % 32
    int P0, int Q0, //RESIZE(resized)
    int H, int W,   //INPUT
    int pt, int pb, int pl, int pr, const int tcount)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos >= tcount) return;

    const int q_idx = pos % Q;
    int idx = pos / Q;
    const int p_idx = idx % P;
    idx /= P;
    const int c_idx = idx % 3;
    const int b_idx = idx / 3;

    if (q_idx < Q0 && p_idx < P0)
    {
        if (H == P0 && W == Q0) {
            int s_idx = b_idx * H * W * 3 + p_idx * W * 3 + q_idx * 3 + 2 - c_idx;
            output[pos + pt * Q + pl] = input[s_idx] / 255.f;
        }
        else {
            const float h1r = area_pixel_compute_source_index(rheight, p_idx);
            const int h1 = h1r;
            const int h1p = (h1 < H - 1) ? 1 : 0;
            const float h1lambda = h1r - h1;
            const float h0lambda = 1.f - h1lambda;

            const float w1r = area_pixel_compute_source_index(rwidth, q_idx);
            const int w1 = w1r;
            const int w1p = (w1 < W - 1) ? 1 : 0;
            const float w1lambda = w1r - w1;
            const float w0lambda = 1.f - w1lambda;

            int base = b_idx * H * W * 3 + h1 * W * 3 + w1 * 3 + 2 - c_idx;

            float val = h0lambda * (w0lambda * input[base] + w1lambda * input[base + w1p * 3]) +
                h1lambda * (w0lambda * input[base + h1p * W * 3] + w1lambda * input[base + h1p * W * 3 + w1p * 3]);
            output[pos + pt * Q + pl] = val / 255.f;
        }
    }
    else {
        if (pl <= q_idx && q_idx < Q0 + pl && pt <= p_idx && p_idx < P0 + pt)
            output[b_idx * P * Q * 3 + c_idx * P * Q + ((P0 - 1 + pt) - p_idx) * Q + ((Q0 - 1 + pl) - q_idx)] = 114.f / 255.f;
        else
            output[pos] = 114.f / 255.f;
    }
}

void preprocess_cu_3(float* output, unsigned char* input, int batchSize, int P, int Q, int P0, int Q0, int H, int W, int pt, int pb, int pl, int pr, hipStream_t stream)
{
    int tcount = batchSize * P * Q * 3;
    int block = 512;
    int grid = (tcount - 1) / block + 1;

    float rheight = float(H) / P0;
    float rwidth = float(W) / Q0;

    kernel_preprocess_3 << <grid, block, 0, stream >> > (output, input, rheight, rwidth, P, Q, P0, Q0, H, W, pt, pb, pl, pr, tcount);
}